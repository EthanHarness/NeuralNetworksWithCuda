﻿#include "NeuralNetwork.h"
#include "CMatrix.cuh"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <chrono>
#include <stdexcept>
#include "hip/hip_runtime.h"
#include ""

void CudaVNonCuda();
std::vector<std::pair<CMatrix, CMatrix>> readTestData();
std::vector<std::pair<CMatrix, CMatrix>> readTrainingData();

int main() {
    //Takes in mnist training sets
    std::vector<std::pair<CMatrix, CMatrix>> testData = readTestData();
    std::vector<std::pair<CMatrix, CMatrix>> trainData = readTrainingData();
    std::cout << "Train Data samples: " << trainData.size() << "\n";
    std::cout << "Test Data samples: " << testData.size() << "\n";

    /*
    Initializes network strucutre. 
    784 (28x28) neurons in layer 1 with 784x10 weights total (CUDA IS GOATED AT THIS)
    10 neurons layer 2 with again 10x10 weights total
    Final layer holds output information. We are choosing 1 out of 10 outputs
    */
    int layer1 = testData[0].first.height;
    const int layer2 = 16;
    const int layer3 = 12;
    const int layer4 = 10;
    const int networkSize = 4;
    const int epochs = 100;
    const int batchSize = 32;
    const double learningRate = .05;
    int networkStructure[] = {layer1, layer2, layer3, layer4};
    NeuralNetwork network = NeuralNetwork(networkStructure, networkSize);
    network.stochasticGradDescent(trainData, epochs, batchSize, learningRate, testData);
    //CudaVNonCuda();
    return 0;
}

//Temporary helper function to test variuos CMat functions
void CudaVNonCuda() {
    //Creates and sets a bunch of CMatrix's (Mainly for testing purposes)
    const int iterations = 1;
    const int matrix_scale_factor = 512;

    std::function<double(int, int)> foo = [](int x, int y) {
        return 1;
    };

    //This does a bunch of Matrix multiplications.
    for(int i = 1; i < iterations*matrix_scale_factor; i+=matrix_scale_factor) {
        CMatrix m1 = createCMatrix(i*matrix_scale_factor, i*matrix_scale_factor);
        CMatrix m2 = createCMatrix(i*matrix_scale_factor, i*matrix_scale_factor);
        setCMatrix(foo, m1);
        setCMatrix(foo, m2);
        std::cout << "Size of Matrix 1 is : " << m1.height << "x" << m1.width << std::endl;

        CMatrix m3 = multiply_cuda(m1,m2);
        std::cout << "Size of Matrix 2 is : " << m3.height << "x" << m3.width << std::endl;

        CMatrix m4 = CMatrixMultiply(m1,m2);
        std::cout << "Size of Matrix 3 is : " << m4.height << "x" << m4.width << std::endl;

        //printCMatrix(m1);
        //printCMatrix(m2);
        //printCMatrix(m3);
        printCMatrix(m4);

        freeCMatrix(m1);
        freeCMatrix(m2);
        freeCMatrix(m3);
   }
}

//Reads in test data for our NN to store
std::vector<std::pair<CMatrix, CMatrix>> readTestData() {
    std::ifstream file("data/mnist_test.csv");

    if (!file.is_open()) {
        throw std::runtime_error("Error: File could not be opened.");
    }

    std::string line;
    std::vector<std::pair<CMatrix, CMatrix>> testData;
    
    //Need to consume the first line since its just header information
    std::getline(file, line);

    const int limit = 100;
    int count = 0;
    while (std::getline(file, line) && (count < limit || limit == -1)) {
        count++;
        std::stringstream ss(line);
        std::string value;

        std::vector<double> row;
        while (std::getline(ss, value, ',')) {
            double dValue = std::stod(value);
            row.push_back(dValue);
        }

        int firstValue = row[0];
        std::function<double(int, int)> firstValueFunc;
        firstValueFunc = [firstValue](int x, int y) {
            if (x == firstValue) {
                return 1.0;
            }
            return 0.0;
        };
        CMatrix expectedOutput = createCMatrix(10, 1);
        setCMatrix(firstValueFunc, expectedOutput);

        CMatrix testingDataCMatrix = createCMatrix(row.size()-1, 1);
        row.erase(row.begin());
        
        std::function<double(int, int)> foo;
        foo = [row](int x, int y) {
            return (row[x]/255.0);
        };
        setCMatrix(foo, testingDataCMatrix);

        testData.push_back(std::make_pair(testingDataCMatrix, expectedOutput));
    }

    file.close();
    return testData;
}

//Reads in training data for our NN to store
std::vector<std::pair<CMatrix, CMatrix>> readTrainingData() {
    std::ifstream file("data/mnist_train.csv");

    if (!file.is_open()) {
        throw std::runtime_error("Error: File could not be opened.");
    }

    std::string line;
    std::vector<std::pair<CMatrix, CMatrix>> testData;
    
    //Need to consume the first line since its just header information
    std::getline(file, line);

    const int limit = 1000;
    int count = 0;
    while (std::getline(file, line) && (count < limit || limit == -1)) {
        count++;
        std::stringstream ss(line);
        std::string value;

        std::vector<double> row;
        while (std::getline(ss, value, ',')) {
            double dValue = std::stod(value);
            row.push_back(dValue);
        }

        int firstValue = row[0];
        std::function<double(int, int)> firstValueFunc;
        firstValueFunc = [firstValue](int x, int y) {
            if (x == firstValue) {
                return 1.0;
            }
            return 0.0;
        };
        CMatrix expectedOutput = createCMatrix(10, 1);
        setCMatrix(firstValueFunc, expectedOutput);

        CMatrix testingDataCMatrix = createCMatrix(row.size()-1, 1);
        row.erase(row.begin());

        std::function<double(int, int)> foo;
        foo = [row](int x, int y) {
            return (row[x]/255.0);
        };
        setCMatrix(foo, testingDataCMatrix);

        testData.push_back(std::make_pair(testingDataCMatrix, expectedOutput));
    }

    file.close();
    return testData;
}